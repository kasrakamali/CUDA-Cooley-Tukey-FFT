#include "hip/hip_runtime.h"
#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// parameters
#define PI 3.14159265

// functions
__global__ void fft2(float* xr, float* xi, int steps) {
    int L = 1;
    double wr;
    double wi;
    double xrt1;
    double xit1;
    double br[2];
    double bi[2];
    //__shared__ float xr[2048];
    //__shared__ float xi[2048];

    int zero;
    int one;
    int q;
    int idx = tx + bx * 1024 + by * 32768 * 1024;

    /*xr[2 * tx] = x_r_d[2 * tx + bx * 2048 + by * 32768 * 2048];
    xi[2 * tx] = x_i_d[2 * tx + bx * 2048 + by * 32768 * 2048];

    xr[2 * tx + 1] = x_r_d[2 * tx + 1 + bx * 2048 + by * 32768 * 2048];
    xi[2 * tx + 1] = x_i_d[2 * tx + 1 + bx * 2048 + by * 32768 * 2048];

    __syncthreads();*/

    for (int i = 0; i < steps; ++i) {
        q = idx % L;
        zero = (idx / L) * (2 * L) + q;
        one = (idx / L) * (2 * L) + q + 1 * L;
        
        wr = cos(q * 2 * PI / (2 * L));
        //wi = -sin(q * 2 * PI / (2 * L));
        wi = -sqrt(1 - wr * wr);

        xrt1 = xr[one] * wr - xi[one] * wi;
        xit1 = xi[one] * wr + xr[one] * wi;


        br[0] = xr[zero] + xrt1;
        bi[0] = xi[zero] + xit1;

        br[1] = xr[zero] - xrt1;
        bi[1] = xi[zero] - xit1;


        xr[zero] = br[0];
        xi[zero] = bi[0];

        xr[one] = br[1];
        xi[one] = bi[1];

        L = L * 2;
        __syncthreads();
    }
    
    /*x_r_d[2 * tx + bx * 2048 + by * 32768 * 2048] = xr[2 * tx];
    x_i_d[2 * tx + bx * 2048 + by * 32768 * 2048] = xi[2 * tx];

    x_r_d[2 * tx + 1 + bx * 2048 + by * 32768 * 2048] = xr[2 * tx + 1];
    x_i_d[2 * tx + 1 + bx * 2048 + by * 32768 * 2048] = xi[2 * tx + 1];*/

}


__global__ void onestepfft2(float* xr, float* xi, int step) {
    int L = 1 << step;
    double wr;
    double wi;
    double xrt1;
    double xit1;
    double br[2];
    double bi[2];

    int zero;
    int one;
    int q;
    int idx = tx + bx * 1024 + by * 32768 * 1024;

    q = idx % L;
    zero = (idx / L) * (2 * L) + q;
    one = (idx / L) * (2 * L) + q + 1 * L;

    wr = cos(q * 2 * PI / (2 * L));
    //wi = -sin(q * 2 * PI / (2 * L));
    wi = -sqrt(1 - wr * wr);

    xrt1 = xr[one] * wr - xi[one] * wi;
    xit1 = xi[one] * wr + xr[one] * wi;


    br[0] = xr[zero] + xrt1;
    bi[0] = xi[zero] + xit1;

    br[1] = xr[zero] - xrt1;
    bi[1] = xi[zero] - xit1;


    xr[zero] = br[0];
    xi[zero] = bi[0];

    xr[one] = br[1];
    xi[one] = bi[1];

}


__global__ void bitrev2(float* xr, float* xi, int bits) {
    float tmpr;
    float tmpi;
    long long idx;
    long long revidx;
    idx = tx + bx * 1024 + by * 32768 * 1024;
    revidx = 0;

    for (int j = 0; j < bits / 2; ++j) {
        revidx |= (idx & (1 << j)) << (bits - 1 - 2 * j);
        revidx |= (idx & (1 << (bits - 1 - j))) >> (bits - 1 - 2 * j);
    }
    if (bits % 2 == 1) {
        revidx |= idx & (1 << (bits / 2));
    }

    if (revidx > idx) {
        tmpr = xr[revidx];
        tmpi = xi[revidx];

        xr[revidx] = xr[idx];
        xi[revidx] = xi[idx];

        xr[idx] = tmpr;
        xi[idx] = tmpi;
    }
}


__global__ void fft4(float* x_r_d, float* x_i_d, int steps) {
    int L = 1;
    double wr1, wr2, wr3;
    double wi1, wi2;
    double xrt[4];
    double xit[4];
    double br[4];
    double bi[4];
    __shared__ float xr[1024];
    __shared__ float xi[1024];

    int zero;
    int one;
    int two;
    int three;
    int q;
    int idx = tx + bx * 256 + by * 32768 * 256;

    xr[4 * tx] = x_r_d[4 * idx];
    xi[4 * tx] = x_i_d[4 * idx];

    xr[4 * tx + 1] = x_r_d[4 * idx + 1];
    xi[4 * tx + 1] = x_i_d[4 * idx + 1];

    xr[4 * tx + 2] = x_r_d[4 * idx + 2];
    xi[4 * tx + 2] = x_i_d[4 * idx + 2];

    xr[4 * tx + 3] = x_r_d[4 * idx + 3];
    xi[4 * tx + 3] = x_i_d[4 * idx + 3];

    __syncthreads();

    for (int i = 0; i < steps; ++i) {
        q = idx % L;
        zero = (tx / L) * (4 * L) + q;
        one = (tx / L) * (4 * L) + q + 1 * L;
        two = (tx / L) * (4 * L) + q + 2 * L;
        three = (tx / L) * (4 * L) + q + 3 * L;


        xrt[0] = xr[zero];
        xit[0] = xi[zero];

        wr1 = cos(q * 2 * PI / (4 * L));
        wi1 = -sqrt(1 - wr1 * wr1);
        xrt[1] = xr[one] * wr1 - xi[one] * wi1;
        xit[1] = xi[one] * wr1 + xr[one] * wi1;

        wr2 = 2 * wr1 * wr1 - 1;
        wi2 = 2 * wr1 * wi1;
        xrt[2] = xr[two] * wr2 - xi[two] * wi2;
        xit[2] = xi[two] * wr2 + xr[two] * wi2;

        wr3 = wr1 * wr2 - wi1 * wi2;
        wi2 = wi2 * wr1 + wi1 * wr2;
        xrt[3] = xr[three] * wr3 - xi[three] * wi2;
        xit[3] = xi[three] * wr3 + xr[three] * wi2;


        br[0] = xrt[0] + xrt[1] + xrt[2] + xrt[3];
        bi[0] = xit[0] + xit[1] + xit[2] + xit[3];

        br[1] = xrt[0] + xit[1] - xrt[2] - xit[3];
        bi[1] = xit[0] - xrt[1] - xit[2] + xrt[3];

        br[2] = xrt[0] - xrt[1] + xrt[2] - xrt[3];
        bi[2] = xit[0] - xit[1] + xit[2] - xit[3];

        br[3] = xrt[0] - xit[1] - xrt[2] + xit[3];
        bi[3] = xit[0] + xrt[1] - xit[2] - xrt[3];


        xr[zero] = br[0];
        xi[zero] = bi[0];

        xr[one] = br[1];
        xi[one] = bi[1];

        xr[two] = br[2];
        xi[two] = bi[2];

        xr[three] = br[3];
        xi[three] = bi[3];

        L = L * 4;
        __syncthreads();

    }

    x_r_d[4 * idx] = xr[4 * tx];
    x_i_d[4 * idx] = xi[4 * tx];

    x_r_d[4 * idx + 1] = xr[4 * tx + 1];
    x_i_d[4 * idx + 1] = xi[4 * tx + 1];

    x_r_d[4 * idx + 2] = xr[4 * tx + 2];
    x_i_d[4 * idx + 2] = xi[4 * tx + 2];

    x_r_d[4 * idx + 3] = xr[4 * tx + 3];
    x_i_d[4 * idx + 3] = xi[4 * tx + 3];

}


__global__ void onestepfft4(float* xr, float* xi, int step) {
    int L = 1 << step;
    double wr1, wr2, wr3;
    double wi1, wi2;
    double xrt[4];
    double xit[4];
    double br[4];
    double bi[4];

    int zero;
    int one;
    int two;
    int three;
    int q;
    int idx = tx + bx * 256 + by * 32768 * 256;

    q = idx % L;
    zero = (idx / L) * (4 * L) + q;
    one = (idx / L) * (4 * L) + q + 1 * L;
    two = (idx / L) * (4 * L) + q + 2 * L;
    three = (idx / L) * (4 * L) + q + 3 * L;


    xrt[0] = xr[zero];
    xit[0] = xi[zero];

    wr1 = cos(q * 2 * PI / (4 * L));
    wi1 = -sqrt(1 - wr1 * wr1);
    xrt[1] = xr[one] * wr1 - xi[one] * wi1;
    xit[1] = xi[one] * wr1 + xr[one] * wi1;

    wr2 = 2 * wr1 * wr1 - 1;
    wi2 = 2 * wr1 * wi1;
    xrt[2] = xr[two] * wr2 - xi[two] * wi2;
    xit[2] = xi[two] * wr2 + xr[two] * wi2;

    wr3 = wr1 * wr2 - wi1 * wi2;
    wi2 = wi2 * wr1 + wi1 * wr2;
    xrt[3] = xr[three] * wr3 - xi[three] * wi2;
    xit[3] = xi[three] * wr3 + xr[three] * wi2;


    br[0] = xrt[0] + xrt[1] + xrt[2] + xrt[3];
    bi[0] = xit[0] + xit[1] + xit[2] + xit[3];

    br[1] = xrt[0] + xit[1] - xrt[2] - xit[3];
    bi[1] = xit[0] - xrt[1] - xit[2] + xrt[3];

    br[2] = xrt[0] - xrt[1] + xrt[2] - xrt[3];
    bi[2] = xit[0] - xit[1] + xit[2] - xit[3];

    br[3] = xrt[0] - xit[1] - xrt[2] + xit[3];
    bi[3] = xit[0] + xrt[1] - xit[2] - xrt[3];


    xr[zero] = br[0];
    xi[zero] = bi[0];

    xr[one] = br[1];
    xi[one] = bi[1];

    xr[two] = br[2];
    xi[two] = bi[2];

    xr[three] = br[3];
    xi[three] = bi[3];

}


__global__ void bitrev4(float* xr, float* xi, int bits) {
    float tmpr;
    float tmpi;
    long long idx;
    long long revidx;
    idx = tx + bx * 1024 + by * 32768 * 1024;
    revidx = 0;
    
    for (int j = 0; j < bits / 2; ++j) {
        revidx |= (idx & (3 << (2 * j))) << (2 * (bits - 1 - 2 * j));
        revidx |= (idx & (3 << (2 * (bits - 1 - j)))) >> (2 * (bits - 1 - 2 * j));
    }
    if (bits % 2 == 1) {
        revidx |= idx & (3 << (2 * (bits / 2)));
    }
    
    if (revidx > idx) {
        tmpr = xr[revidx];
        tmpi = xi[revidx];

        xr[revidx] = xr[idx];
        xi[revidx] = xi[idx];

        xr[idx] = tmpr;
        xi[idx] = tmpi;
    }
}


__global__ void mixedbitrev42(float* xr, float* xi, float* xr_cpy, float* xi_cpy, int bits) {
    long long idx;
    long long revidx;
    idx = tx + bx * 1024 + by * 32768 * 1024;
    revidx = 0;

    for (int j = 0; j < bits / 2; ++j) {
        revidx |= (idx & (3 << (2 * j))) << (2 * (bits - 1 - 2 * j));
        revidx |= (idx & (3 << (2 * (bits - 1 - j)))) >> (2 * (bits - 1 - 2 * j));
    }
    if (bits % 2 == 1) {
        revidx |= idx & (3 << (2 * (bits / 2)));
    }

    revidx = revidx << 1;
    revidx |= (idx & (1 << (bits * 2))) >> (bits * 2);

    xr[idx] = xr_cpy[revidx];
    xi[idx] = xi_cpy[revidx];
}


__global__ void arraycpy(float* xr, float* xi, float* xr_cpy, float* xi_cpy) {
    long long idx;
    idx = tx + bx * 1024 + by * 32768 * 1024;

    xr_cpy[idx] = xr[idx];
    xi_cpy[idx] = xi[idx];
}


//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
    float* xr_cpy;
    float* xi_cpy;


    dim3 dimGrid1(N / 1024 > 65535 ? 32768 : (N < 1024 ? 1 : N / 1024), N / 1024 > 65535 ? N / 1024 / 32768 : 1);
    dim3 dimBlock1(N > 1024 ? 1024 : N);

    dim3 dimGrid2(N / 2048 > 65535 ? 32768 : (N < 2048 ? 1 : N / 2048), N / 2048 > 65535 ? N / 2048 / 32768 : 1);
    dim3 dimBlock2(N / 2 > 1024 ? 1024 : N / 2);

    dim3 dimGrid3(N / 1024 > 65535 ? 32768 : (N < 1024 ? 1 : N / 1024), N / 1024 > 65535 ? N / 1024 / 32768 : 1);
    dim3 dimBlock3(N / 4 > 256 ? 256 : N / 4);


    if (M % 2 == 0) {
        bitrev4 <<<dimGrid1, dimBlock1>>> (x_r_d, x_i_d, M / 2);

        if (M < 12) {
            fft4 <<<dimGrid3, dimBlock3>>> (x_r_d, x_i_d, M / 2);
        }
        else {
            fft4 <<<dimGrid3, dimBlock3>>> (x_r_d, x_i_d, 10 / 2);
            for (unsigned int i = 10; i < M; i = i + 2) {
                onestepfft4 <<<dimGrid3, dimBlock3>>> (x_r_d, x_i_d, i);
            }
        }
    }
    else {
        HANDLE_ERROR(hipMalloc((void**)&xr_cpy, N * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void**)&xi_cpy, N * sizeof(float)));
        arraycpy <<<dimGrid1, dimBlock1>>> (x_r_d, x_i_d, xr_cpy, xi_cpy);
        mixedbitrev42 <<<dimGrid1, dimBlock1>>> (x_r_d, x_i_d, xr_cpy, xi_cpy, (M - 1) / 2);
        HANDLE_ERROR(hipFree(xr_cpy));
        HANDLE_ERROR(hipFree(xi_cpy));

        if (M < 10) {
            fft4 <<<dimGrid3, dimBlock3>>> (x_r_d, x_i_d, M / 2);
            onestepfft2 <<<dimGrid2, dimBlock2>>> (x_r_d, x_i_d, M - 1);
        }
        else {
            fft4 <<<dimGrid3, dimBlock3>>> (x_r_d, x_i_d, 10 / 2);
            for (unsigned int i = 10; i < M - 1; i = i + 2) {
                onestepfft4 <<<dimGrid3, dimBlock3>>> (x_r_d, x_i_d, i);
            }
            onestepfft2 <<<dimGrid2, dimBlock2>>> (x_r_d, x_i_d, M - 1);
        }

    }

}
